#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include ""
#include <iostream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <stdio.h>

using namespace std;
using namespace thrust;

const int DUOMENU_KIEKIS = 19;
const int MAX_STRING_ILGIS = 250;

struct Zaidejas
{
public:
	char vardas[MAX_STRING_ILGIS];
	int rungtynes;
	double taskai;
	int pavadinimoIlgis;
};

typedef struct Zaidejas Zaidejas;

struct sumavimas {
	//GPU ir CPU funkcija
	__host__ __device__ Zaidejas operator ()(Zaidejas accumulator, Zaidejas item) 
	{
		int ilgisPavadinimo = 0;
		for (int i = accumulator.pavadinimoIlgis; i < accumulator.pavadinimoIlgis + item.pavadinimoIlgis; i++)
		{
			accumulator.vardas[i] = item.vardas[ilgisPavadinimo];
			ilgisPavadinimo++;
		}
		accumulator.pavadinimoIlgis = accumulator.pavadinimoIlgis + item.pavadinimoIlgis;
		accumulator.rungtynes = accumulator.rungtynes + item.rungtynes;
		accumulator.taskai = accumulator.taskai + item.taskai;

		return accumulator;
	}
};

//skaitymo is failo funkcija
host_vector<Zaidejas> skaityti();

//i faila rasymas
void spausdinti(char vardas[], int rungtynes, double taskai, int pavadinimoIlgis);

int main() {

	//skaitom duomenis
	host_vector<Zaidejas> zaidejai = skaityti();

	//kopinam i� CPU i GPU arba CPU i CPU
	//device_vector<Zaidejas> zaidejai_GPU(zaidejai);//gpu
	host_vector<Zaidejas> zaidejai_GPU(zaidejai);//cpu

	Zaidejas temp;
	temp.pavadinimoIlgis = 0;
	temp.rungtynes = 0;
	temp.taskai = 0.0;

	//funktoriaus panaudojimas
	auto res = reduce(zaidejai_GPU.begin(), zaidejai_GPU.end(), temp, sumavimas());

	for (int j = 0; j < res.pavadinimoIlgis; j++) {
		cout << res.vardas[j];
	}

	cout << " - -> " << res.rungtynes << " - -> " << res.taskai << endl;
	spausdinti(res.vardas, res.rungtynes, res.taskai, res.pavadinimoIlgis);
	return 0;
}

host_vector<Zaidejas> skaityti()
{

	host_vector<Zaidejas> zaidejai(0);
	string nVardas;
	int nRungtynes;
	double nTaskai;
	ifstream fd("duomenys.txt");

	for (int i = 0; i < DUOMENU_KIEKIS; i++) {

		fd >> nVardas >> nRungtynes >> nTaskai;

		Zaidejas temp;
		string vardas = nVardas;
		int rungtynes = nRungtynes;
		double taskai = nTaskai;

		strcpy(temp.vardas, vardas.c_str());
		temp.rungtynes = rungtynes;
		temp.taskai = taskai;
		temp.pavadinimoIlgis = strlen(temp.vardas);

		zaidejai.push_back(temp);
	}
	fd.close();

	return zaidejai;
}

void spausdinti(char vardas[], int rungtynes, double taskai, int pavadinimoIlgis)
{
	ofstream Write;
	Write.open("rezai.txt");

	for (int i = 0; i < pavadinimoIlgis; i++) {
		Write << vardas[i];
	}
	Write << " - -> " << rungtynes << " - -> " << taskai << endl;

	Write.close();
}

