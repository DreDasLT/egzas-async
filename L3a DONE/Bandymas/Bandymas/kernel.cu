#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>

using namespace std;

const int GIJU_SKAICIUS = 4;
const int DUOMENU_KIEKIS = 19;
const int VIETA = 10;
const int REZULTATU_VIETA = VIETA * (DUOMENU_KIEKIS / GIJU_SKAICIUS);

struct Zaidejas {
public:

	char vardas[REZULTATU_VIETA];
	int rungtynes;
	double taskai;

	//CPU
	//konstruktorius
	__host__ Zaidejas() {
		memset(vardas, ' ', REZULTATU_VIETA);
		rungtynes = 0;
		taskai = 0.0;
	};


	//GPU
	//konstruktorius su parametrais
	__device__ Zaidejas(char vardas[], int rungtynes, double taskai) {
		for (int i = 0; i < REZULTATU_VIETA; i++) {
			this->vardas[i] = vardas[i];
		}
		this->rungtynes = rungtynes;
		this->taskai = taskai;
	}
};

hipError_t vykdyti(Zaidejas* duomenys, Zaidejas* rezultatai);

__global__ void sumavimas(Zaidejas* zaidejai, Zaidejas* rezultatai);

void skaityti(Zaidejas* zaidejai);

void spausdinti(Zaidejas* duomenys, Zaidejas* rezultatai);


int main()
{
	Zaidejas* zaidejai = new Zaidejas[DUOMENU_KIEKIS];
	skaityti(zaidejai);

	/*
	for (int i = 0; i < DUOMENU_KIEKIS; i++) {
		cout << i << zaidejai[i].pavadinimas << " " << zaidejai[i].metai << " " << zaidejai[i].litrai << endl;
	}
	*/

	Zaidejas* rezultatai = new Zaidejas[GIJU_SKAICIUS];
	hipError_t cudaStatus = vykdyti(zaidejai, rezultatai);

	spausdinti(zaidejai, rezultatai);

	/*
	cout << "REZAI" << endl;
	for (int i = 0; i < GIJU_SKAICIUS; i++) {
		for (int j = 0; j < REZULTATU_VIETA; j++) {
			cout << rezultatai[i].pavadinimas[j];
		}
		cout << " ->" << rezultatai[i].metai << "-->" << rezultatai[i].litrai << endl;
	}
	*/

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida kai GPU vykde");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	
	delete[] zaidejai;
	delete[] rezultatai;
	return 0;
}



hipError_t vykdyti(Zaidejas* duomenys, Zaidejas* rezultatai)
{
	hipError_t cudaStatus;

	//kintamieji skirti GPU darbui
	Zaidejas* gpu_rezultatai = new Zaidejas[GIJU_SKAICIUS];
	Zaidejas* gpu_duomenys = new Zaidejas[DUOMENU_KIEKIS];

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Nerastas GPU");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& gpu_duomenys, DUOMENU_KIEKIS * sizeof(Zaidejas));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida isskiriant vieta!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& gpu_rezultatai, GIJU_SKAICIUS * sizeof(Zaidejas));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	//kopijuojam is vienos atminties i kita
	cudaStatus = hipMemcpy(gpu_duomenys, duomenys, DUOMENU_KIEKIS * sizeof(Zaidejas), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida kopijuojant is CPU i GPU!");
		goto Error;
	}
	cudaStatus = hipMemcpy(gpu_rezultatai, rezultatai, GIJU_SKAICIUS * sizeof(Zaidejas), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida kopijuojant is CPU i GPU!");
		goto Error;
	}

	//lygiagreti dalis
	sumavimas << <1, GIJU_SKAICIUS >> > (gpu_duomenys, gpu_rezultatai);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Ivykusios klaidos: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida grazinta laukiant giju pabaigos: %d\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(rezultatai, gpu_rezultatai, GIJU_SKAICIUS * sizeof(Zaidejas), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Klaida kopijuojant is GPU i CPU\n");
		goto Error;
	}
Error:
	hipFree(gpu_duomenys);
	hipFree(gpu_rezultatai);

	return cudaStatus;
}

//pagrindinis sumavimas
__global__ void sumavimas(Zaidejas* zaidejai, Zaidejas* rezultatai)
{
	int poKiekImti = DUOMENU_KIEKIS / GIJU_SKAICIUS;
	int gijosNr = threadIdx.x;
	int sumaIki = 0;
	int sumaNuo = 0;
	int z = 0;
	int rungtynes = 0;
	double taskai = 0.0;
	char vardai[REZULTATU_VIETA];

	if (gijosNr == 0) {
		sumaNuo = 0;
		sumaIki = poKiekImti;
	}
	else if (gijosNr == GIJU_SKAICIUS - 1) {
		sumaNuo = gijosNr * poKiekImti;
		sumaIki = DUOMENU_KIEKIS + 1;
	}
	else {
		sumaNuo = gijosNr * poKiekImti;
		sumaIki = (gijosNr + 1) * poKiekImti;
	}

	//printf(" %d--->%d\n", sumaNuo, sumaIki);

	for (int i = sumaNuo; i < sumaIki; i++) {
		rungtynes += (int)zaidejai[i].rungtynes;
		taskai += (double)zaidejai[i].taskai;

		for (int p = 0; p < REZULTATU_VIETA; p++) {
			if (zaidejai[i].vardas[p] != ' ' && zaidejai[i].vardas[p] != '\0') {
				vardai[z] = zaidejai[i].vardas[p];
				z++;
			}
		}
	}

	rezultatai[gijosNr] = Zaidejas(vardai, rungtynes, taskai);
}

//skaitymas is failo
void skaityti(Zaidejas* zaidejai) {

	string zVardas;
	int zRungtynes;
	double zTaskai;
	ifstream fd("duomenys.txt");
	for (int i = 0; i < DUOMENU_KIEKIS; i++) {

		fd >> zVardas >> zRungtynes >> zTaskai;
		strcpy(zaidejai[i].vardas, zVardas.c_str());
		zaidejai[i].taskai = zTaskai;
		zaidejai[i].rungtynes = zRungtynes;
	}
	fd.close();
}

//visko isvedimas i faila
void spausdinti(Zaidejas* duomenys, Zaidejas* rezultatai) {

	ofstream Write("rezultatai.txt");
	Write << "-------->-------Pradiniai duomenys--------<-------" << endl;
	Write << endl;
	Write << "Vardas                                                                 Rungtynes   Taskai" << endl;

	for (int i = 0; i < DUOMENU_KIEKIS; i++) {
		Write << i + 1 << ".) ";
		for (int j = 0; j < REZULTATU_VIETA; j++) {
			Write << duomenys[i].vardas[j];
		}
		Write << "- ->" << duomenys[i].rungtynes << "- ->" << duomenys[i].taskai << endl;
	}
	
	Write << endl;
	Write << endl;
	Write << endl;

	Write << "-------->-------REZULTATAI--------<-------" << endl;
	Write << endl;
	Write << "Vardas                                   Rungtynes   Taskai" << endl;

	for (int i = 0; i < GIJU_SKAICIUS; i++) {
		Write << i + 1 << ".) ";
		for (int j = 0; j < REZULTATU_VIETA; j++) {
			Write << rezultatai[i].vardas[j];
		}
		Write << "- ->" << rezultatai[i].rungtynes << "- ->" << rezultatai[i].taskai << endl;
	}
	Write.close();

}


